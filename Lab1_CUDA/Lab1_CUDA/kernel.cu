#include "hip/hip_runtime.h"
﻿#include <iostream>
#include <fstream>
#include <vector>
#include <hip/hip_runtime.h>

__global__ void dotProductKernel(float* a, float* b, float* result, int n) {
    float tmp = 0.0f;
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Каждый поток считает произведение своих элементов
    if (idx < n) {
        tmp = a[idx] * b[idx];
    }

    // Используем атомарное добавление для точного суммирования
    atomicAdd(result, tmp);
}

float dotProductCUDA(float* a, float* b, int n) {
    float* d_a, * d_b, * d_result;
    float h_result = 0.0f;

    hipMalloc((void**)&d_a, n * sizeof(float));
    hipMalloc((void**)&d_b, n * sizeof(float));
    hipMalloc((void**)&d_result, sizeof(float));

    hipMemcpy(d_a, a, n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, n * sizeof(float), hipMemcpyHostToDevice);

    // Инициализация результата
    hipMemset(d_result, 0, sizeof(float));

    int blockSize = 50;
    int numBlocks = (n + blockSize) / blockSize;

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    // Запуск ядра
    dotProductKernel << <numBlocks, blockSize >> > (d_a, d_b, d_result, n); // numBlocks по сути количество ядер, но его менять не стоит в данной реализации количество ядер вычисляется по количеству блоков, поэтому меняйте размер блоков это BlockSize

    hipMemcpy(&h_result, d_result, sizeof(float), hipMemcpyDeviceToHost);

    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_result);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    std::cout << "Время выполнения: " << milliseconds << " мс" << std::endl;
    return h_result;
}

bool readVectorFromFile(const std::string& filename, std::vector<float>& vec) {
    std::ifstream file(filename);
    if (!file.is_open()) {
        std::cerr << "Ошибка открытия файла: " << filename << std::endl;
        return false;
    }

    float value;
    while (file >> value) {
        vec.push_back(value);
    }
    file.close();
    return true;
}

int main() {
    setlocale(LC_ALL, "");
    std::vector<float> a, b;

    if (!readVectorFromFile("v1.txt", a) || !readVectorFromFile("v2.txt", b)) {
        return -1;
    }

    if (a.size() != b.size()) {
        std::cerr << "Размеры векторов не совпадают!" << std::endl;
        return -1;
    }

    float result = dotProductCUDA(a.data(), b.data(), a.size());
    std::cout << "Скалярное произведение: " << result << std::endl;

    return 0;
}
